
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>

extern "C" void getlasterror_ (int *tag) {

  hipError_t status;
  status = hipGetLastError();
  printf("Last error: %d %d \n",*tag,status);
  printf("Cuda error: %s \n", hipGetErrorString( status) );
  fflush(stdout);
  return;
}
