// This routine is used by SMS.
// This routine returns a pointer to an exchange variable.
// The pointers table in this routine must contain all the variables to be exchanged.
// Eventually PPP will generate the pointers table.
// Currently (April 2012) the pointers table is hard coded for FIM.
// Author:  Jacques Middlecoff
// Date:    August 2012

// Variables that appear here must be malloc'd in one of the copytoGPU routines.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
extern "C" void sms_getpointer (char *varname, void **ptr, int *status, int string_len) {

extern int   *D_prox;
extern float *D_nprox;
extern float *D_proxs;
extern float *D_lat;
extern float *D_lon;
extern float *D_delp_lo;
extern float *D_delp;
extern float *D_r_plus;
extern float *D_r_mnus;
extern float *D_ptdcy;
extern float *D_area;
extern float *D_cs;
extern float *D_sn;
extern float *D_sidevec_c;
extern float *D_sidevec_e;
extern float *D_sideln;
extern float *D_rprox_ln;
extern float *D_corio;
extern float *D_deg_lat;
extern float *D_deg_lon;
extern float *D_rarea;
extern float *D_rsideln;
extern float *D_actual;
extern float *D_work_edg;
extern float *D_us3d;
extern float *D_vs3d;
extern float *D_dp3d;
extern float *D_tr3d;
extern float *D_mp3d;
extern float *D_pr3d;
extern float *D_fields;
extern float *D_thko;
extern float *D_thkn;
extern float *D_flp;
extern float *D_fln;
extern float *D_workb;
extern float *D_exsmo3d;
extern float *D_fld;
extern float *D_cumufx;
extern float *D_dpinit;
extern int   *D_dpfinl;
extern float *D_massfx;
extern float *D_trcr_edg;
extern float *D_tracr;
extern float *D_trcr_lo;
extern float *D_g3p;

struct pointers {
  char *string;
  void *D_;
};

struct pointers table[] = {{"prox"     , D_prox     },
                           {"nprox"    , D_nprox    },
                           {"proxs"    , D_proxs    },
                           {"lat"      , D_lat      },
                           {"lon"      , D_lon      },
                           {"delp_lo"  , D_delp_lo  },
                           {"delp"     , D_delp     },
                           {"r_plus"   , D_r_plus   },
                           {"r_mnus"   , D_r_mnus   },
                           {"ptdcy"    , D_ptdcy    },
                           {"area"     , D_area     },
                           {"cs"       , D_cs       },
                           {"sn"       , D_sn       },
                           {"sidevec_c", D_sidevec_c},
                           {"sidevec_e", D_sidevec_e},
                           {"sideln"   , D_sideln   },
                           {"rprox_ln" , D_rprox_ln },
                           {"corio"    , D_corio    },
                           {"deg_lat"  , D_deg_lat  },
                           {"deg_lon"  , D_deg_lon  },
                           {"rarea"    , D_rarea    },
                           {"rsideln"  , D_rsideln  },
                           {"actual"   , D_actual   },
                           {"work_edg" , D_work_edg },
                           {"us3d"     , D_us3d     },
                           {"vs3d"     , D_vs3d     },
                           {"dp3d"     , D_dp3d     },
                           {"tr3d"     , D_tr3d     },
                           {"mp3d"     , D_mp3d     },
                           {"pr3d"     , D_pr3d     },
                           {"fields"   , D_fields   },
                           {"thko"     , D_thko     },
                           {"thkn"     , D_thkn     },
                           {"flp"      , D_flp      },
                           {"fln"      , D_fln      },
                           {"workb"    , D_workb    },
                           {"exsmo3d"  , D_exsmo3d  },
                           {"fld"      , D_fld      },
                           {"cumufx"   , D_cumufx   },
                           {"dpinit"   , D_dpinit   },
                           {"dpfinl"   , D_dpfinl   },
                           {"massfx"   , D_massfx   },
                           {"trcr_edg" , D_trcr_edg },
                           {"tracr"    , D_tracr    },
                           {"trcr_lo"  , D_trcr_lo  },
                           {"g3p"      , D_g3p      }
                          };

 int i;
 const int numelem = sizeof (table) / sizeof (struct pointers);

 *status=0;

  for (i = 0; i < numelem; ++i) {
//  printf("JFM: %d %s %d %d %s %d \n",i,varname,string_len,strlen(table[i].string),table[i].string,status);
    if( (string_len == strlen(table[i].string)) && (strncmp(varname,table[i].string,string_len) == 0) ) { 
      *ptr = table[i].D_ ;
      return;
    }
  }
  printf("Error in SMS_GetPointer.cu: VarName out of range %s %d \n",varname,string_len);
  printf("Make sure all exchange variables are in the pointers table in SMS_GetPointer.cu \n");
  *status=-8003;
  return;
}
