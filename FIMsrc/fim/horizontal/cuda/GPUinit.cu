// Routine to initialize the GPU
// Author:  Jacques Middlecoff
// Date:  September 2010 
// For Fortran this routine does nothing except return error=0.

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cutil.h>
#include "ftocmacros.h"

extern "C" void gpuinit_ (int *npes,int *me,int *error) {

int argc=2;
char *argv0[]= {"","-device=0"};
char *argv1[]= {"","-device=1"};
hipDeviceProp_t deviceProp;

*error = 0;

#if CUDART_VERSION < 2020
#error "This CUDART version does not support mapped memory!\n"
#endif

// Get properties and verify device 0 supports mapped memory
*error = hipGetDeviceProperties(&deviceProp, 0);
if(*error != hipSuccess) {
  printf("GPUinit.cu: hipGetDeviceProperties error %d \n",*error);
  return;
}
if(!deviceProp.canMapHostMemory) {
  printf("GPUinit.cu: Device %d cannot map host memory!\n", 0);
  *error = -88;
  return;
}

if(*me%2 == 0)
{
  CUT_DEVICE_INIT(argc, argv0);
  printf("Processor %d %s \n",*me,argv0[1]);
} 
else
{
  CUT_DEVICE_INIT(argc, argv1);
  printf("Processor %d %s \n",*me,argv1[1]);
}

// set the device flags for mapping host memory
*error = hipSetDeviceFlags(hipDeviceMapHost);
if(*error != hipSuccess) {
  printf("GPUinit.cu: hipSetDeviceFlags error %d \n",*error);
  return;
}
}
